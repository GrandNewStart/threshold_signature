#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "recon_key_cuda.h"

__global__ void lagrangeInterpolationKernel_int(int* xs, int* ys, int* results, int numShares, int mod) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < numShares) {
        int xi = xs[idx];
        int yi = ys[idx];
        int li = 1;

        // Compute Lagrange basis polynomial L_i(0)
        for (int j = 0; j < numShares; ++j) {
            if (j == idx) continue;

            int xj = xs[j];
            int numerator = xj;
            int denominator = (xj - xi + mod) % mod;

            // Compute modular inverse of denominator
            int denomInv = 1;
            for (int k = 1; k < mod; ++k) { // Brute-force modular inverse
                if ((denominator * k) % mod == 1) {
                    denomInv = k;
                    break;
                }
            }

            li = (li * numerator % mod * denomInv % mod) % mod;
        }

        // Compute contribution to the final secret
        results[idx] = (yi * li % mod);
    }
}

int reconstructKey_int_CUDA(const std::vector<SHARE_INT>& shares, int mod) {
    int numShares = shares.size();

    // Allocate memory on the device
    int* d_xs;
    int* d_ys;
    int* d_results;
    hipMalloc(&d_xs, numShares * sizeof(int));
    hipMalloc(&d_ys, numShares * sizeof(int));
    hipMalloc(&d_results, numShares * sizeof(int));

    // Copy shares to the device
    std::vector<int> xs(numShares), ys(numShares);
    for (int i = 0; i < numShares; ++i) {
        xs[i] = shares[i].x;
        ys[i] = shares[i].y;
    }
    hipMemcpy(d_xs, xs.data(), numShares * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, ys.data(), numShares * sizeof(int), hipMemcpyHostToDevice);

    // Start timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Configure and launch the kernel
    int blockSize = 256;
    int gridSize = (numShares + blockSize - 1) / blockSize;
    hipEventRecord(start);
    lagrangeInterpolationKernel_int << <gridSize, blockSize >> > (d_xs, d_ys, d_results, numShares, mod);
    hipEventRecord(stop);

    // Wait for the GPU to finish
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the timing result
    std::cout << "[reconstructKey_int_CUDA] " << milliseconds << " ms" << std::endl;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Copy results back to the host
    std::vector<int> results(numShares);
    hipMemcpy(results.data(), d_results, numShares * sizeof(int), hipMemcpyDeviceToHost);

    // Aggregate contributions to reconstruct the secret
    int secret = 0;
    for (int r : results) {
        secret = (secret + r) % mod;
    }

    // Free device memory
    hipFree(d_xs);
    hipFree(d_ys);
    hipFree(d_results);

    return secret;
}

__device__ unsigned long long modInverse_long_device(unsigned long long a, unsigned long long mod) {
    unsigned long long m0 = mod, t, q;
    unsigned long long x0 = 0, x1 = 1;

    if (mod == 1) return 0;

    while (a > 1) {
        q = a / mod;
        t = mod;

        mod = a % mod;
        a = t;
        t = x0;

        x0 = x1 - q * x0;
        x1 = t;
    }

    if (x1 < 0)
        x1 += m0;

    return x1;
}

__global__ void lagrangeInterpolationKernel_long(
    int numShares,
    unsigned long long* xs,
    unsigned long long* ys,
    unsigned long long* results,
    unsigned long long mod
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numShares) {
        unsigned long long lambda = 1;

        for (int j = 0; j < numShares; ++j) {
            if (j == idx) continue;

            // Compute denominator
            unsigned long long denom = (xs[j] - xs[idx] + mod) % mod;
            if (denom == 0) {
                printf("Thread %d: Denom is zero for j = %d, xs[j] = %llu, xs[idx] = %llu\n", idx, j, xs[j], xs[idx]);
                return; // Skip if invalid denom
            }

            // Compute modular inverse
            unsigned long long denomInv = modInverse_long_device(denom, mod);
            if (denomInv == 0) {
                printf("Thread %d: modInverse failed for denom = %llu\n", idx, denom);
                return; // Skip if modular inverse fails
            }

            // Update lambda
            unsigned long long oldLambda = lambda;
            lambda = (lambda % mod * xs[j] % mod * denomInv % mod) % mod;

            // Log intermediate values
            //printf("Thread %d: j = %d, denom = %llu, denomInv = %llu, oldLambda = %llu, lambda = %llu\n",
            //    idx, j, denom, denomInv, oldLambda, lambda);
        }

        // Compute result
        results[idx] = (lambda * ys[idx]) % mod;

        // Log final result
        //printf("Thread %d: final lambda = %llu, result = %llu\n", idx, lambda, results[idx]);
    }
}

unsigned long long reconstructKey_long_CUDA(const std::vector<SHARE_LONG>& shares, unsigned long long order) {
    int numShares = shares.size();
    unsigned long long* h_xs, * h_ys, * h_results;
    unsigned long long* d_xs, * d_ys, * d_results;

    h_xs = new unsigned long long[numShares];
    h_ys = new unsigned long long[numShares];
    h_results = new unsigned long long[numShares];

    for (int i = 0; i < numShares; ++i) {
        h_xs[i] = shares[i].x;
        h_ys[i] = shares[i].y;
    }

    hipMalloc(&d_xs, numShares * sizeof(unsigned long long));
    hipMalloc(&d_ys, numShares * sizeof(unsigned long long));
    hipMalloc(&d_results, numShares * sizeof(unsigned long long));

    hipMemcpy(d_xs, h_xs, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, h_ys, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);


    // Start timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = 256;
    int gridSize = (numShares + blockSize - 1) / blockSize;
    hipEventRecord(start);
    lagrangeInterpolationKernel_long << <gridSize, blockSize >> > (numShares, d_xs, d_ys, d_results, order);
    hipEventRecord(stop);

    // Wait for the GPU to finish
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the timing result
    std::cout << "[reconstructKey_int_CUDA] " << milliseconds << " ms" << std::endl;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_results, d_results, numShares * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    unsigned long long secret = 0;
    for (int i = 0; i < numShares; ++i) {
        secret = (secret + h_results[i]) % order;
    }

    delete[] h_xs;
    delete[] h_ys;
    delete[] h_results;
    hipFree(d_xs);
    hipFree(d_ys);
    hipFree(d_results);

    return secret;
}

BIGNUM* reconstructKey_BIGNUM_CUDA(const std::vector<SHARE_BIGNUM>& shares, const BIGNUM* mod) {
    int numShares = shares.size();
    unsigned long long* h_xs, * h_ys, * h_results;
    unsigned long long* d_xs, * d_ys, * d_results;

    h_xs = new unsigned long long[numShares];
    h_ys = new unsigned long long[numShares];
    h_results = new unsigned long long[numShares];

    for (int i = 0; i < numShares; ++i) {
        h_xs[i] = BN_get_word(shares[i].x);
        h_ys[i] = BN_get_word(shares[i].y);
    }

    hipMalloc(&d_xs, numShares * sizeof(unsigned long long));
    hipMalloc(&d_ys, numShares * sizeof(unsigned long long));
    hipMalloc(&d_results, numShares * sizeof(unsigned long long));

    hipMemcpy(d_xs, h_xs, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, h_ys, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);


    // Start timing using CUDA events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int blockSize = 256;
    int gridSize = (numShares + blockSize - 1) / blockSize;
    hipEventRecord(start);
    lagrangeInterpolationKernel_long << <gridSize, blockSize >> > (numShares, d_xs, d_ys, d_results, BN_get_word(mod));
    hipEventRecord(stop);

    // Wait for the GPU to finish
    hipEventSynchronize(stop);

    // Calculate the elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the timing result
    std::cout << "[reconstructKey_int_CUDA] " << milliseconds << " ms" << std::endl;

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipMemcpy(h_results, d_results, numShares * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    unsigned long long secret = 0;
    for (int i = 0; i < numShares; ++i) {
        secret = (secret + h_results[i]) % BN_get_word(mod);
    }

    BIGNUM* reconsestructed = BN_new();
    BN_set_word(reconsestructed, secret);

    delete[] h_xs;
    delete[] h_ys;
    delete[] h_results;
    hipFree(d_xs);
    hipFree(d_ys);
    hipFree(d_results);

    return reconsestructed;
}