﻿#include "hip/hip_runtime.h"
#include ""

#include "common.h"
#include "gen_key.h"
#include "gen_shares.h"
#include "gen_shares_cuda.h"
#include "recon_key.h"
#include "recon_key_cuda.h"

bool showLogs = false;

void test_int() {
    try {
        // Step 1: Generate a random key
        int secret = generateKey_int(1, 100);
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        int n = 400;
        int t = 200;
        int mod = 101;
        auto shares = generateShares_int(secret, n, t, mod);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        // Step 3: Reconstruct the key from the shares
        std::vector<SHARE_INT> selectedShares(shares.begin(), shares.begin() + t);
        int reconstructed = reconstructKey_int(selectedShares, mod);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        // Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_int]            SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_int]            FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "[test_int] Error: " << e.what() << std::endl;
    }
}

void test_int_cuda() {
    try {
        // Step 1: Generate a random key
        int secret = generateKey_int(1, 100);
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        int n = 100, t = 50, mod = 101;
        auto shares = generateShares_int_CUDA(secret, n, t, mod);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        // Reconstruct the secret using CUDA
        int reconstructed = reconstructKey_int_CUDA(shares, mod);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        // Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_int_cuda]       SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_int_cuda]       FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

void test_long() {
    try {
        unsigned long long order = 1000000007;
        unsigned long long secret = generateKey_long(order);
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        int n = 5, t = 3;
        std::vector<SHARE_LONG> shares = generateShares_long(secret, n, t, order);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        unsigned long long reconstructed = reconstructKey_long(shares, order);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        // Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_long]           SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_long]           FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

void test_long_cuda() {
    try {
        unsigned long long order = 1000000007;
        unsigned long long secret = generateKey_long(order);
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        int n = 5, t = 3;
        std::vector<SHARE_LONG> shares = generateShares_long_CUDA(secret, n, t, order);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        unsigned long long reconstructed = reconstructKey_long_CUDA(shares, order);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        // Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_long_cuda]      SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_long_cuda]      FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

void test_BIGNUM() {
    try {
        // Generate key pair
        BIGNUM* secret = generateKey_BIGNUM();
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        // Generate 100 shares with a threshold of 50
        int n = 100, t = 50;
        std::vector<SHARE_BIGNUM> shares = generateShares_BIGNUM(secret, n, t);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        // Reconstruct the original key pair from shares
        BIGNUM* reconstructed = reconstructKey_BIGNUM(shares);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        if (BN_cmp(secret, reconstructed) != 0) {
            std::cout << "[test_BIGNUM]         SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_BIGNUM]         FAILED X" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.x); // Free private share
            BN_free(share.y); // Free public key
        }

        BN_free(secret);
        BN_free(reconstructed);
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

void test_BIGNUM_cuda() {
    try {
        BIGNUM* mod = BN_new();
        BN_set_word(mod, 1000000007);

        // Generate key pair
        BIGNUM* secret = generateKey_BIGNUM();
        if (showLogs) {
            std::cout << "Original Secret Key: " << secret << std::endl;
        }

        // Generate 100 shares with a threshold of 50
        int n = 100, t = 50;
        std::vector<SHARE_BIGNUM> shares = generateShares_BIGNUM_CUDA(secret, n, t, mod);
        if (showLogs) {
            std::cout << "Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }

        // Reconstruct the original key pair from shares
        BIGNUM* reconstructed = reconstructKey_BIGNUM_CUDA(shares, mod);
        if (showLogs) {
            std::cout << "Reconstructed Secret Key: " << reconstructed << std::endl;
        }

        if (BN_cmp(secret, reconstructed) != 0) {
            std::cout << "[test_BIGNUM_cuda]    SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_BIGNUM_cuda]    FAILED X" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.x); // Free private share
            BN_free(share.y); // Free public key
        }

        BN_free(secret);
        BN_free(reconstructed);
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

int main() {
    test_int();
    test_int_cuda();
    test_long();
    test_long_cuda();
    test_BIGNUM();
    test_BIGNUM_cuda();
    return 0;
}
