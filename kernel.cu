﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <vector>
#include <openssl/bn.h>
#include <openssl/ec.h>
#include "threshold_crypto.h"
#include "threshold_crypto_cuda.h"

void printKeyPair(std::pair<BIGNUM*, BIGNUM*> keyPair) {
    char* privHex1 = BN_bn2hex(keyPair.first);
    std::cout << "Private Key: " << privHex1 << std::endl;
    OPENSSL_free(privHex1);

    char* pubHex1 = BN_bn2hex(keyPair.second);
    std::cout << "Public Key: " << pubHex1 << std::endl;
    OPENSSL_free(pubHex1);
}

void printShares(std::vector<std::pair<BIGNUM*, BIGNUM*>> shares) {
    for (auto& share : shares) {
        std::cout << "share: " << share.first << ", " << share.second << std::endl;
    }
}

void test_aggregate_key() {
    try {
        // Generate key pair
        std::pair<BIGNUM*, BIGNUM*> keyPair = generateKeyPair();
        printKeyPair(keyPair);

        // Generate 100 shares with a threshold of 50
        int n = 5, t = 3;
        std::vector<std::pair<BIGNUM*, BIGNUM*>> shares = generateShares(keyPair.first, n, t);
        printShares(shares);

        // Reconstruct the original key pair from shares
        std::pair<BIGNUM*, BIGNUM*> reconstructed = reconstructKeyPair(shares);
        printKeyPair(reconstructed);

        if (BN_cmp(keyPair.first, reconstructed.first) != 0) {
            std::cerr << "Private keys do not match!" << std::endl;
        }
        else {
            std::cout << "Private keys match!" << std::endl;
        }
        if (BN_cmp(keyPair.second, reconstructed.second) != 0) {
            std::cerr << "Public keys do not match!" << std::endl;
        }
        else {
            std::cout << "Public keys match!" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.first); // Free private share
            BN_free(share.second); // Free public key
        }

        BN_free(keyPair.first);
        BN_free(keyPair.second);

        BN_free(reconstructed.first);
        BN_free(reconstructed.second);
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }

}

void test_aggregate_key_cuda() {
    try {
        // Generate key pair
        std::pair<BIGNUM*, BIGNUM*> keyPair = generateKeyPair();
        printKeyPair(keyPair);

        // Number of shares and threshold
        int n = 5, t = 3;

        // Get the elliptic curve group and modulus
        EC_GROUP* group = EC_GROUP_new_by_curve_name(NID_X9_62_prime256v1);
        if (!group) {
            throw std::runtime_error("Failed to create EC_GROUP for P-256");
        }
        const BIGNUM* mod = EC_GROUP_get0_order(group);
        if (!mod) {
            throw std::runtime_error("Failed to get curve group order");
        }

        // Generate shares using CUDA
        std::vector<std::pair<BIGNUM*, BIGNUM*>> shares = generateSharesCUDA(keyPair.first, n, t, mod);
        printShares(shares);

        // Convert shares for reconstruction
        std::vector<std::pair<BIGNUM*, BIGNUM*>> sharesVector;
        for (const auto& share : shares) {
            sharesVector.push_back({ share.first, share.second });
        }

        // Reconstruct the original key pair using CUDA
        std::pair<BIGNUM*, BIGNUM*> reconstructed = reconstructKeyCUDA(sharesVector, mod);
        printKeyPair(reconstructed);

        // Validate the reconstructed keys
        if (BN_cmp(keyPair.first, reconstructed.first) != 0) {
            std::cerr << "Private keys do not match!" << std::endl;
        }
        else {
            std::cout << "Private keys match!" << std::endl;
        }
        if (BN_cmp(keyPair.second, reconstructed.second) != 0) {
            std::cerr << "Public keys do not match!" << std::endl;
        }
        else {
            std::cout << "Public keys match!" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.first);  // Free private share
            BN_free(share.second); // Free public key
        }

        BN_free(keyPair.first);
        BN_free(keyPair.second);

        BN_free(reconstructed.first);
        BN_free(reconstructed.second);

        EC_GROUP_free(group);
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

int main()
{
    //test_aggregate_key();
    test_aggregate_key_cuda();
    return 0;
}
