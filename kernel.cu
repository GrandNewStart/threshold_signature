﻿#include <chrono> 

#include "hip/hip_runtime.h"
#include ""

#include "common.h"
#include "gen_key.h"
#include "gen_shares.h"
#include "gen_shares_cuda.h"
#include "recon_key.h"
#include "recon_key_cuda.h"

bool showLogs = false;
bool showTimes = true;
int n = 1000;
int t = 200;

void test_int() {
    try {
        // Step 1: Generate a random key
        int secret = generateKey_int(1, 100);
        if (showLogs) {
            std::cout << "[test_int] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        int mod = 101;
        auto start = std::chrono::high_resolution_clock::now();
        auto shares = generateShares_int(secret, n, t, mod);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_int] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_int] share generation time: " << duration.count() << " ms" << std::endl;
        }

        // Step 3: Reconstruct the key from the shares
        std::vector<SHARE_INT> selectedShares(shares.begin(), shares.begin() + t);
        start = std::chrono::high_resolution_clock::now();
        int reconstructed = reconstructKey_int(selectedShares, mod);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_int] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_int] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_int] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_int] FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "[test_int] Error: " << e.what() << std::endl;
    }
}


void test_int_cuda() {
    try {
        // Step 1: Generate a random key
        int secret = generateKey_int(1, 100);
        if (showLogs) {
            std::cout << "[test_int_cuda] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        int mod = 101;
        auto start = std::chrono::high_resolution_clock::now();
        std::vector<SHARE_INT> shares = generateShares_int_CUDA(secret, n, t, mod);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_int_cuda] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_int_cuda] share generation time: " << duration.count() << " ms" << std::endl;
        }
 
        // Step 3: Reconstruct the key from the shares
        start = std::chrono::high_resolution_clock::now();
        int reconstructed = reconstructKey_int_CUDA(shares, mod);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_int_cuda] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_int_cuda] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_int_cuda] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_int_cuda] FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "[test_int_cuda] Error: " << e.what() << std::endl;
    }
}

void test_long() {
    try {
        unsigned long long order = 1000000007;

        // Step 1: Generate a random key
        unsigned long long secret = generateKey_long(order);
        if (showLogs) {
            std::cout << "[test_long] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        auto start = std::chrono::high_resolution_clock::now();
        std::vector<SHARE_LONG> shares = generateShares_long(secret, n, t, order);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_long] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_long] share generation time: " << duration.count() << " ms" << std::endl;
        }

        // Step 3: Reconstruct the key from the shares
        start = std::chrono::high_resolution_clock::now();
        unsigned long long reconstructed = reconstructKey_long(shares, order);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_long] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_long] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_long] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_long] FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "[test_long] Error: " << e.what() << std::endl;
    }
}

void test_long_cuda() {
    try {
        unsigned long long order = 1000000007;

        // Step 1: Generate a random key
        unsigned long long secret = generateKey_long(order);
        if (showLogs) {
            std::cout << "[test_long_cuda] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        auto start = std::chrono::high_resolution_clock::now();
        std::vector<SHARE_LONG> shares = generateShares_long_CUDA(secret, n, t, order);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_long_cuda] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_long_cuda] share generation time: " << duration.count() << " ms" << std::endl;
        }

        // Step 3: Reconstruct the key from the shares
        start = std::chrono::high_resolution_clock::now();
        unsigned long long reconstructed = reconstructKey_long_CUDA(shares, order);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_long_cuda] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_long_cuda] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (reconstructed == secret) {
            std::cout << "[test_long_cuda] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_long_cuda] FAILED X" << std::endl;
        }
    }
    catch (const std::exception& e) {
        std::cerr << "Error: " << e.what() << std::endl;
    }
}

void test_BIGNUM() {
    try {
        // Step 1: Generate a random key
        BIGNUM* secret = generateKey_BIGNUM();
        if (showLogs) {
            std::cout << "[test_BIGNUM] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        auto start = std::chrono::high_resolution_clock::now();
        std::vector<SHARE_BIGNUM> shares = generateShares_BIGNUM(secret, n, t);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_BIGNUM] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_BIGNUM] share generation time: " << duration.count() << " ms" << std::endl;
        }

        // Step 3: Reconstruct the key from the shares
        start = std::chrono::high_resolution_clock::now();
        BIGNUM* reconstructed = reconstructKey_BIGNUM(shares);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_BIGNUM] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_BIGNUM] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (BN_cmp(secret, reconstructed) != 0) {
            std::cout << "[test_BIGNUM] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_BIGNUM] FAILED X" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.x); // Free private share
            BN_free(share.y); // Free public key
        }

        BN_free(secret);
        BN_free(reconstructed);
    }
    catch (const std::exception& e) {
        std::cerr << "[test_BIGNUM] Error: " << e.what() << std::endl;
    }
}

void test_BIGNUM_cuda() {
    try {
        BIGNUM* mod = BN_new();
        BN_set_word(mod, 1000000007);

        // Step 1: Generate a random key
        BIGNUM* secret = generateKey_BIGNUM();
        if (showLogs) {
            std::cout << "[test_BIGNUM_cuda] Original Secret Key: " << secret << std::endl;
        }

        // Step 2: Split the key into shares
        auto start = std::chrono::high_resolution_clock::now();
        std::vector<SHARE_BIGNUM> shares = generateShares_BIGNUM_CUDA(secret, n, t, mod);
        auto end = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double, std::milli> duration = end - start;
        if (showLogs) {
            std::cout << "[test_BIGNUM_cuda] Generated Shares:" << std::endl;
            for (const auto& share : shares) {
                std::cout << "x: " << share.x << ", y: " << share.y << std::endl;
            }
        }
        if (showTimes) {
            std::cout << "[test_BIGNUM_cuda] share generation time: " << duration.count() << " ms" << std::endl;
        }

        // Step 3: Reconstruct the key from the shares
        start = std::chrono::high_resolution_clock::now();
        BIGNUM* reconstructed = reconstructKey_BIGNUM_CUDA(shares, mod);
        end = std::chrono::high_resolution_clock::now();
        duration = end - start;
        if (showLogs) {
            std::cout << "[test_BIGNUM_cuda] Reconstructed Secret Key: " << reconstructed << std::endl;
        }
        if (showTimes) {
            std::cout << "[test_BIGNUM_cuda] key reconstruction time: " << duration.count() << " ms" << std::endl;
        }

        // Step 4: Verify correctness
        if (BN_cmp(secret, reconstructed) != 0) {
            std::cout << "[test_BIGNUM_cuda] SUCCEESS V" << std::endl;
        }
        else {
            std::cout << "[test_BIGNUM_cuda] FAILED X" << std::endl;
        }

        // Clean up
        for (auto& share : shares) {
            BN_free(share.x); // Free private share
            BN_free(share.y); // Free public key
        }

        BN_free(secret);
        BN_free(reconstructed);
    }
    catch (const std::exception& e) {
        std::cerr << "[test_BIGNUM_cuda] Error: " << e.what() << std::endl;
    }
}

int main() {
    test_int();
    std::cout << "====================================================" << std::endl;
    test_int_cuda();

    std::cout << std::endl;

    test_long();
    std::cout << "====================================================" << std::endl;
    test_long_cuda();

    std::cout << std::endl;

    test_BIGNUM();
    std::cout << "====================================================" << std::endl;
    test_BIGNUM_cuda();
    return 0;
}
