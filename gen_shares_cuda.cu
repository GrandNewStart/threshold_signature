#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include "common.h"
#include "gen_coef.h"

__global__ void evaluatePolynomial_int_CUDA(
    int* coefficients, 
    int secret, 
    int* xs, 
    int* ys, 
    int degree, 
    int mod, 
    int n
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < n) {
        int x = xs[idx];
        int y = secret;
        int powerOfX = 1;

        // Evaluate the polynomial at x
        for (int i = 0; i < degree; ++i) {
            powerOfX = (powerOfX * x) % mod; // Compute x^i mod p
            y = (y + coefficients[i] * powerOfX) % mod;
        }

        ys[idx] = y; // Store the result
    }
}

std::vector<SHARE_INT> generateShares_int_CUDA(int secret, int n, int t, int mod) {
    if (t > n) {
        throw std::invalid_argument("Threshold t cannot be greater than number of shares n");
    }

    // Generate random coefficients for the polynomial on the host
    auto coefficients = generateCoefficients_int(t - 1, mod);

    // Allocate memory on the device
    int* d_coefficients;
    int* d_xs;
    int* d_ys;
    hipMalloc(&d_coefficients, (t - 1) * sizeof(int));
    hipMalloc(&d_xs, n * sizeof(int));
    hipMalloc(&d_ys, n * sizeof(int));

    // Copy coefficients to the device
    hipMemcpy(d_coefficients, coefficients.data(), (t - 1) * sizeof(int), hipMemcpyHostToDevice);

    // Generate x-coordinates on the host
    std::vector<int> xs(n);
    for (int i = 0; i < n; ++i) {
        xs[i] = i + 1; // x = 1, 2, ..., n
    }

    // Copy x-coordinates to the device
    hipMemcpy(d_xs, xs.data(), n * sizeof(int), hipMemcpyHostToDevice);

    // Configure and launch the kernel
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    evaluatePolynomial_int_CUDA << <gridSize, blockSize >> > (d_coefficients, secret, d_xs, d_ys, t - 1, mod, n);

    // Copy results back to the host
    std::vector<int> ys(n);
    hipMemcpy(ys.data(), d_ys, n * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_coefficients);
    hipFree(d_xs);
    hipFree(d_ys);

    // Combine x- and y-coordinates into shares
    std::vector<SHARE_INT> shares;
    for (int i = 0; i < n; ++i) {
        shares.push_back({ xs[i], ys[i] });
    }

    return shares;
}

__global__ void evaluatePolynomial_long_CUDA(
    int n,
    int t,
    unsigned long long* coefficients,
    unsigned long long* xs,
    unsigned long long* ys,
    unsigned long long mod
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        unsigned long long x = xs[idx];
        unsigned long long result = 0;
        unsigned long long xPow = 1;

        for (int i = 0; i < t; ++i) {
            result = (result + coefficients[i] * xPow) % mod;
            xPow = (xPow * x) % mod;
        }

        ys[idx] = result;
    }
}


std::vector<SHARE_LONG> generateShares_long_CUDA(unsigned long long privateKey, int n, int t, unsigned long long order) {
    auto coefficients = generateCoefficients_long(privateKey, t, order);
    unsigned long long* h_coefficients, * h_xs, * h_ys;
    unsigned long long* d_coefficients, * d_xs, * d_ys;

    h_coefficients = new unsigned long long[t];
    for (int i = 0; i < t; ++i) {
        h_coefficients[i] = coefficients[i];
    }

    // Allocate memory
    hipMalloc(&d_coefficients, t * sizeof(unsigned long long));
    hipMalloc(&d_xs, n * sizeof(unsigned long long));
    hipMalloc(&d_ys, n * sizeof(unsigned long long));

    // Transfer coefficients to the device
    hipMemcpy(d_coefficients, h_coefficients, t * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Initialize x values on host
    h_xs = new unsigned long long[n];
    for (int i = 0; i < n; ++i) {
        h_xs[i] = i + 1; // x values are 1 to n
    }
    hipMemcpy(d_xs, h_xs, n * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Launch kernel to evaluate polynomial
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    evaluatePolynomial_long_CUDA << <gridSize, blockSize >> > (n, t, d_coefficients, d_xs, d_ys, order);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    // Copy results back to host
    h_ys = new unsigned long long[n];
    hipMemcpy(h_ys, d_ys, n * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Generate shares
    std::vector<SHARE_LONG> shares;
    for (int i = 0; i < n; ++i) {
        shares.push_back({ h_xs[i], h_ys[i] });
    }

    // Free memory
    delete[] h_coefficients;
    delete[] h_xs;
    delete[] h_ys;
    hipFree(d_coefficients);
    hipFree(d_xs);
    hipFree(d_ys);

    return shares;
}

std::vector<SHARE_BIGNUM> generateShares_BIGNUM_CUDA(const BIGNUM* privateKey, int n, int t, const BIGNUM* mod) {
    std::vector<BIGNUM*> coefficients = generateCoefficients_BIGNUM(privateKey, t); // Assuming generatePolynomial is defined
    unsigned long long* h_coefficients, * h_xs, * h_ys;
    unsigned long long* d_coefficients, * d_xs, * d_ys;

    // Convert coefficients to device-friendly format
    h_coefficients = new unsigned long long[t];
    for (int i = 0; i < t; ++i) {
        h_coefficients[i] = BN_get_word(coefficients[i]);
    }

    // Allocate memory
    hipMalloc(&d_coefficients, t * sizeof(unsigned long long));
    hipMalloc(&d_xs, n * sizeof(unsigned long long));
    hipMalloc(&d_ys, n * sizeof(unsigned long long));

    // Transfer coefficients to the device
    hipMemcpy(d_coefficients, h_coefficients, t * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Initialize x values on host
    h_xs = new unsigned long long[n];
    for (int i = 0; i < n; ++i) {
        h_xs[i] = i + 1; // x values are 1 to n
    }
    hipMemcpy(d_xs, h_xs, n * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Launch kernel to evaluate polynomial
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    evaluatePolynomial_long_CUDA << <gridSize, blockSize >> > (n, t, d_coefficients, d_xs, d_ys, BN_get_word(mod));

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << std::endl;
    }

    // Copy results back to host
    h_ys = new unsigned long long[n];
    hipMemcpy(h_ys, d_ys, n * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Generate shares and public keys
    std::vector<SHARE_BIGNUM> shares;
    for (int i = 0; i < n; ++i) {
        BIGNUM* x = BN_new();
        BIGNUM* y = BN_new();
        BN_set_word(x, h_xs[i]);
        BN_set_word(y, h_ys[i]);
        shares.push_back({ x, y});
    }

    // Free memory
    delete[] h_coefficients;
    delete[] h_xs;
    delete[] h_ys;
    hipFree(d_coefficients);
    hipFree(d_xs);
    hipFree(d_ys);

    return shares;
}
