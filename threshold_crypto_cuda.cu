#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <vector>
#include <openssl/bn.h>
#include <openssl/evp.h>
#include <openssl/ec.h>
#include <stdexcept>

#include "threshold_crypto_cuda.h"
#include "threshold_crypto.h"

// Kernel to evaluate the polynomial at a given x for all shares
__global__ void evaluatePolynomialKernel(
    int n, 
    int t, 
    unsigned long long* coefficients, 
    unsigned long long* xs, 
    unsigned long long* ys, 
    unsigned long long mod
) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < n) {
        unsigned long long x = xs[idx];
        unsigned long long result = 0;
        unsigned long long xPow = 1;

        for (int i = 0; i < t; ++i) {
            result = (result + coefficients[i] * xPow) % mod;
            xPow = (xPow * x) % mod;
        }

        ys[idx] = result;
    }
}

std::vector<BIGNUM*> generatePolynomial(const BIGNUM* secret, int t) {
    std::vector<BIGNUM*> coefficients;
    coefficients.push_back(BN_dup(secret)); // Constant term is the secret

    for (int i = 1; i < t; ++i) {
        BIGNUM* coeff = BN_new();
        if (!BN_rand_range(coeff, BN_get0_nist_prime_256())) { // Generate coefficients < order
            throw std::runtime_error("Failed to generate random coefficients");
        }
        coefficients.push_back(coeff);
    }

    return coefficients;
}


// Function to split the private key into shares
std::vector<std::pair<BIGNUM*, BIGNUM*>> generateSharesCUDA(const BIGNUM* privateKey, int n, int t, const BIGNUM* mod) {
    std::vector<BIGNUM*> coefficients = generatePolynomial(privateKey, t); // Assuming generatePolynomial is defined
    unsigned long long* h_coefficients, * h_xs, * h_ys;
    unsigned long long* d_coefficients, * d_xs, * d_ys;

    // Convert coefficients to device-friendly format
    h_coefficients = new unsigned long long[t];
    for (int i = 0; i < t; ++i) {
        h_coefficients[i] = BN_get_word(coefficients[i]);
    }

    // Allocate memory
    hipMalloc(&d_coefficients, t * sizeof(unsigned long long));
    hipMalloc(&d_xs, n * sizeof(unsigned long long));
    hipMalloc(&d_ys, n * sizeof(unsigned long long));

    // Transfer coefficients to the device
    hipMemcpy(d_coefficients, h_coefficients, t * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Initialize x values on host
    h_xs = new unsigned long long[n];
    for (int i = 0; i < n; ++i) {
        h_xs[i] = i + 1; // x values are 1 to n
    }
    hipMemcpy(d_xs, h_xs, n * sizeof(unsigned long long), hipMemcpyHostToDevice);

    // Launch kernel to evaluate polynomial
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    evaluatePolynomialKernel << <gridSize, blockSize >> > (n, t, d_coefficients, d_xs, d_ys, BN_get_word(mod));

    // Copy results back to host
    h_ys = new unsigned long long[n];
    hipMemcpy(h_ys, d_ys, n * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Generate shares and public keys
    std::vector<std::pair<BIGNUM*, BIGNUM*>> shares;
    for (int i = 0; i < n; ++i) {
        BIGNUM* y = BN_new();
        BN_set_word(y, h_ys[i]);
        BIGNUM* publicKey = generatePublicKey(y);
        shares.push_back({ y, publicKey });
    }

    // Free memory
    delete[] h_coefficients;
    delete[] h_xs;
    delete[] h_ys;
    hipFree(d_coefficients);
    hipFree(d_xs);
    hipFree(d_ys);

    return shares;
}



__device__ unsigned long long modInverse(unsigned long long a, unsigned long long mod) {
    unsigned long long m0 = mod, t, q;
    unsigned long long x0 = 0, x1 = 1;

    if (mod == 1) return 0;

    while (a > 1) {
        // q is the quotient
        q = a / mod;
        t = mod;

        // m is the remainder now, process the next step
        mod = a % mod, a = t;
        t = x0;

        // Update x0 and x1
        x0 = x1 - q * x0;
        x1 = t;
    }

    // Make x1 positive
    if (x1 < 0)
        x1 += m0;

    return x1;
}


__global__ void lagrangeInterpolationKernel(int numShares, unsigned long long* xs, unsigned long long* ys, unsigned long long* results, unsigned long long mod) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < numShares) {
        unsigned long long lambda = 1;

        for (int j = 0; j < numShares; ++j) {
            if (j == idx) continue;

            unsigned long long denom = (xs[j] - xs[idx] + mod) % mod;
            denom = modInverse(denom, mod); // Modular inverse

            lambda = (lambda * xs[j] % mod * denom % mod) % mod;
        }

        results[idx] = (lambda * ys[idx]) % mod;
    }
}

// Host function for Lagrange interpolation
std::pair<BIGNUM*, BIGNUM*> reconstructKeyCUDA(const std::vector<std::pair<BIGNUM*, BIGNUM*>>& shares, const BIGNUM* mod) {
    int numShares = shares.size();
    unsigned long long* h_xs, * h_ys, * h_results;
    unsigned long long* d_xs, * d_ys, * d_results;

    h_xs = new unsigned long long[numShares];
    h_ys = new unsigned long long[numShares];
    h_results = new unsigned long long[numShares];

    for (int i = 0; i < numShares; ++i) {
        h_xs[i] = BN_get_word(shares[i].first);
        h_ys[i] = BN_get_word(shares[i].second);
    }

    hipMalloc(&d_xs, numShares * sizeof(unsigned long long));
    hipMalloc(&d_ys, numShares * sizeof(unsigned long long));
    hipMalloc(&d_results, numShares * sizeof(unsigned long long));

    hipMemcpy(d_xs, h_xs, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);
    hipMemcpy(d_ys, h_ys, numShares * sizeof(unsigned long long), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (numShares + blockSize - 1) / blockSize;
    lagrangeInterpolationKernel << <gridSize, blockSize >> > (numShares, d_xs, d_ys, d_results, BN_get_word(mod));

    hipMemcpy(h_results, d_results, numShares * sizeof(unsigned long long), hipMemcpyDeviceToHost);

    unsigned long long secret = 0;
    for (int i = 0; i < numShares; ++i) {
        secret = (secret + h_results[i]) % BN_get_word(mod);
    }

    BIGNUM* reconstructedSecret = BN_new();
    BN_set_word(reconstructedSecret, secret);
    BIGNUM* publicKey = generatePublicKey(reconstructedSecret);

    delete[] h_xs;
    delete[] h_ys;
    delete[] h_results;
    hipFree(d_xs);
    hipFree(d_ys);
    hipFree(d_results);

    return { reconstructedSecret, publicKey };
}